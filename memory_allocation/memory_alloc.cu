
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <unistd.h>

// Memory transfer host -> device -> host is essentially a 3 stage process:
// 1. Device memory allocation. Memory reservation
// 2. Host to device memory transfer.
// 3. Device memory de-allocation.

int main()

{
    int n_devices;
    hipError_t err = hipGetDeviceCount(&n_devices);
    printf("Error %s\n", hipGetErrorString(err));
    printf("N devices %d\n", n_devices);

    // set CUDA device for all subsequent host-thread operations.
    int cur_device = 5;
    hipSetDevice(cur_device);

    float *A_d, *B_d, *C_d;
    size_t size = 5000000 * sizeof(float);
    hipError_t alloc_err_a = hipMalloc(&A_d, size);
    hipError_t alloc_err_b = hipMalloc(&B_d, size);
    hipError_t alloc_err_c = hipMalloc(&C_d, size);
    if (alloc_err_a != hipSuccess)
    {
        printf("Allocation Error A_d %s\n", hipGetErrorString(alloc_err_a));
    }
    if (alloc_err_b != hipSuccess)
    {
        printf("Allocation Error B_d %s\n", hipGetErrorString(alloc_err_b));
    }
    if (alloc_err_c != hipSuccess)
    {
        printf("Allocation Error C_d %s\n", hipGetErrorString(alloc_err_c));
    }
    sleep(10);
    hipFree(A_d);
    hipFree(B_d);
    hipFree(C_d);
    // an attempt of double free!
    hipFree(&A_d);
}