#include "../kernel.cu"
#include "../util.c"

#include <cmath>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

int main() {
  int a_rows = 4;
  int a_cols = 8;
  int b_rows = 8;
  int b_cols = 2;

  float *A_host, *B_host, *C_host;
  float *A_device, *B_device, *C_device;

  int a_size = a_rows * a_cols * sizeof(float);
  int b_size = b_rows * b_cols * sizeof(float);
  int c_size = a_rows * b_cols * sizeof(float);

  A_host = (float *)malloc(a_size);
  B_host = (float *)malloc(b_size);
  C_host = (float *)malloc(c_size);

  // init host data
  init(A_host, a_rows * a_cols);
  init(B_host, b_rows * b_cols);

  hipMalloc((void **)&A_device, a_size);
  hipMalloc((void **)&B_device, b_size);
  hipMalloc((void **)&C_device, c_size);

  hipMemcpy(A_device, A_host, a_size, hipMemcpyHostToDevice);
  hipMemcpy(B_device, B_host, b_size, hipMemcpyHostToDevice);

  dim3 threadsPerBlock(16, 16);

  dim3 numBlocks((b_rows + threadsPerBlock.x - 1) / threadsPerBlock.x,
                 (a_cols + threadsPerBlock.y - 1) / threadsPerBlock.y);

  naiveMatMulKernel<<<numBlocks, threadsPerBlock>>>(
      A_device, B_device, C_device, a_rows, a_cols, b_cols);

  hipMemcpy(C_host, C_device, c_size, hipMemcpyDeviceToHost);

  hipDeviceSynchronize();

  hipFree(A_device);
  hipFree(B_device);
  hipFree(C_device);

  print_tensor(C_host, a_rows * b_cols);

  free(A_host);
  free(B_host);
  free(C_host);

  return 0;
}