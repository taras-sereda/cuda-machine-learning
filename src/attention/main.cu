#include "../kernel.cu"
#include "../util.c"

#include <cmath>

#include <hipblaslt.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>

#include <stdio.h>
#include <stdlib.h>

int main() {
  int a_rows = 4;
  int a_cols = 8;
  int b_rows = 8;
  int b_cols = 2;

  float *A_host, *B_host, *C_host, *D_host;
  float *A_device, *B_device, *C_device, *D_device;

  int a_size = a_rows * a_cols * sizeof(float);
  int b_size = b_rows * b_cols * sizeof(float);
  int c_size = a_rows * b_cols * sizeof(float);
  int d_size = a_rows * b_cols * sizeof(float);

  A_host = (float *)malloc(a_size);
  B_host = (float *)malloc(b_size);
  C_host = (float *)malloc(c_size);
  D_host = (float *)malloc(c_size);

  // init host data
  init(A_host, a_rows * a_cols);
  init(B_host, b_rows * b_cols);

  hipMalloc((void **)&A_device, a_size);
  hipMalloc((void **)&B_device, b_size);
  hipMalloc((void **)&C_device, c_size);
  hipMalloc((void **)&D_device, d_size);

  hipMemcpy(A_device, A_host, a_size, hipMemcpyHostToDevice);
  hipMemcpy(B_device, B_host, b_size, hipMemcpyHostToDevice);

  dim3 threadsPerBlock(16, 16);

  dim3 numBlocks((b_rows + threadsPerBlock.x - 1) / threadsPerBlock.x,
                 (a_cols + threadsPerBlock.y - 1) / threadsPerBlock.y);

  naiveMatMulKernel<<<numBlocks, threadsPerBlock>>>(
      A_device, B_device, C_device, a_rows, a_cols, b_cols);

  hipDeviceSynchronize();
  hipMemcpy(C_host, C_device, c_size, hipMemcpyDeviceToHost);
  print_tensor(C_host, a_rows * b_cols);

  //   hipblasLtMatmul()
  hipblasHandle_t handle;
  hipblasStatus_t status;
  status = hipblasCreate(&handle);

  int version;
  status = cublasGetVersion(handle, &version);
  printf("CUBLAS get version status: %d\n", status);
  printf("cublas version: %d\n", version);

  float alpha = 1.0f;
  float beta = 0.0f;
  hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, a_rows, b_cols, a_cols, &alpha,
              A_device, a_rows, B_device, b_rows, &beta, D_device, a_rows);

  hipMemcpy(D_host, D_device, d_size, hipMemcpyDeviceToHost);

  print_tensor(D_host, a_rows * b_cols);

  hipFree(A_device);
  hipFree(B_device);
  hipFree(C_device);
  hipFree(D_device);

  free(A_host);
  free(B_host);
  free(C_host);
  free(D_host);

  return 0;
}