#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>

#define ANSI_COLOR_RESET "\x1b[0m"
#define ANSI_COLOR_CYAN "\x1B[36m"

int main()
{
  int nDevices;

  hipError_t err = hipGetDeviceCount(&nDevices);
  if (err != hipSuccess)
    printf("%s\n", hipGetErrorString(err));

  printf("\nCUDA Runtime version: %d.%d\n\n", CUDART_VERSION / 1000, (CUDART_VERSION % 100) / 10); // CUDART_VERSION is defined in hip/hip_runtime_api.h

  for (int devIdx = 0; devIdx < nDevices; devIdx++)
  {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, devIdx);
    printf("Device Number: %d\n", devIdx);
    printf("  Device name: %s\n", prop.name);
    printf("  Device compute capability: %d.%d\n", prop.major, prop.minor);

    // Print the UUID
    printf("  UUID: ");
    for (int i = 0; i < 16; ++i)
    {
      printf("%02x", (unsigned char)prop.uuid.bytes[i]);
      if (i % 4 == 3 && i != 15)
        printf("-");
    }
    printf("\n");

    printf("  LUID: %s(undefined on non-Windows)\n", prop.luid);
    printf("  CanMapHostMemory: %d\n", prop.canMapHostMemory); // True if device supports zero-copy memory mapping. When cuda threads can access host memory directly
    printf("  Clock rate (KHz): %d\n", prop.clockRate);
    printf("  Memory clock rate (KHz): %d\n", prop.memoryClockRate);
    printf("  Memory bus width (bits): %d\n", prop.memoryBusWidth);
    printf("  Peak memory bandwidth (GB/s): %f\n", 2.0 * prop.memoryClockRate * (prop.memoryBusWidth / 8) / 1.0e6);

    printf("  Multiprocessor count, aka SMs: %d\n", prop.multiProcessorCount);
    printf("  Max threads per SM: %d\n", prop.maxThreadsPerMultiProcessor);
    printf("  Registers per block: %d (can be less then per SM!)\n", prop.regsPerBlock);
    printf("  Registers per SM: %d\n", prop.regsPerMultiprocessor);
    printf("  Warp Size: %d\n", prop.warpSize);

    printf("  Max grid size: %d %d %d\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
    printf("  Max threads per block: %d\n", prop.maxThreadsPerBlock);
    printf("  %sMax threads dim: %d %d %d%s\n", ANSI_COLOR_CYAN, prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2], ANSI_COLOR_RESET);
  }
}
