#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>

// single precision ax + y kernel
__global__ void saxpy(int n, float a, float* x, float* y) {

	int i = blockIdx.x * blockDim.x + threadIdx.x;
        if (i < n) y[i] = a * x[i] + y[i];
} 


int main(int argc, char** argv) {

        int nDevices; 
        hipGetDeviceCount(&nDevices);  

        printf("Number of GPU devices %d\n", nDevices);

        for (int i = 0; i < nDevices; i++) {
       
            hipDeviceProp_t prop;
            hipGetDeviceProperties(&prop, i);
           
            printf("Device Number: %d\n", i); 
            printf("  Device Name: %s\n", prop.name);
            printf("  Memory Clock Rate (Khz): %d\n", prop.memoryClockRate);
            printf("  Memory Bus Width (bits): %d\n", prop.memoryBusWidth);
            printf("  Peak Memory Bandwidth (GB/s): %f\n",
                    2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
        }	

	int N = 1<<20; // same as 2^20 or around 1 million. 	
        float *x, *y, *d_x, *d_y;
        x = (float*)malloc(N*sizeof(float));
        y = (float*)malloc(N*sizeof(float));
        
        hipMalloc(&d_x, N*sizeof(float));
        hipMalloc(&d_y, N*sizeof(float));
       
        for (int i = 0; i < N; i++) {
            x[i] = 1.0f;
            y[i] = 2.0f;
	}
	hipMemcpy(d_x, x, N*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_y, y, N*sizeof(float), hipMemcpyHostToDevice);

	saxpy<<<4096, 256>>>(N, 2.0f, d_x, d_y);

	hipMemcpy(y, d_y, N*sizeof(float), hipMemcpyDeviceToHost);

        float maxError = 0.0f; //no suffix defines double
        float vectorSum = 0.0f;
        for (int i = 0; i < N; i++) {
	   maxError = max(maxError, abs(4.0f - y[i]));
           vectorSum += y[i];
        }
        printf("Max error: %f\n", maxError);
        printf("Sum of all vector elements: %f\n", vectorSum);

	// Free device memory
	hipFree(d_x);
	hipFree(d_y);
	// Free host memory
        free(x);
        free(y);
}
