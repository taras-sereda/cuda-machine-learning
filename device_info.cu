#include <stdio.h> 

#define ANSI_COLOR_RESET "\x1b[0m"
#define ANSI_COLOR_CYAN  "\x1B[36m"

int main() {
  int nDevices;

  hipGetDeviceCount(&nDevices);
  for (int devIdx = 0; devIdx < nDevices; devIdx++) {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, devIdx);
    printf("Device Number: %d\n", devIdx);
    printf("  Device name: %s\n", prop.name);
    printf("  Device compute capability: %d.%d\n", prop.major, prop.minor);
    printf("  LUID: %s\n", prop.luid);
    printf("  CanMapHostMemory: %d\n", prop.canMapHostMemory);
    printf("  Clock Rate (KHz): %d\n",
           prop.clockRate);
    printf("  Memory Clock Rate (KHz): %d\n",
           prop.memoryClockRate);
    printf("  Memory Bus Width (bits): %d\n",
           prop.memoryBusWidth);
    printf("  Peak Memory Bandwidth (GB/s): %f\n",
           2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);

    // https://developer.nvidia.com/blog/cuda-pro-tip-the-fast-way-to-query-device-properties/
    printf("  %s%s%s \n", ANSI_COLOR_CYAN, "Querying CUDA Attributes via hipDeviceGetAttribute", ANSI_COLOR_RESET);

    int memoryClockRate;
    hipDeviceGetAttribute(&memoryClockRate, hipDeviceAttributeMemoryClockRate, devIdx);
    printf("  Memory Clock Rate (KHz): %d\n\n",
           memoryClockRate);
  }
}
